#include "hip/hip_runtime.h"

#include "csp_transform_cu.h"
#include <stdio.h>

__global__ void rgba_to_yuv420_kernel(hipTextureObject_t tex_obj, char* y, char* u, char* v, int w, int h, int y_pitch, int uv_pitch)
{
    int vetex_y = blockIdx.y * blockDim.y + threadIdx.y;
    int vetex_x = blockIdx.x * blockDim.x + threadIdx.x;
    if (vetex_x >= w || vetex_y >= h) { return; }

    float tex_x = (vetex_x + 0.5f) / w;
    float tex_y = 1.0f - (vetex_y + 0.5f) / h;
    float4 color = tex2D<float4>(tex_obj, tex_x, tex_y);

    int i = vetex_y*y_pitch + vetex_x;
    //y[i] = (0.1826f*color.x + 0.6142f*color.y + 0.0620f*color.z)*256 + 16;
    //y[i] = (306*color.x + 601*color.y + 117*color.z)*255.0f/1024.0f;
    y[i] = 66*color.x + 129*color.y + 25*color.z + 16; //BT601 limited range
    //y[i] = 77*color.x + 150*color.y + 29*color.z; //BT601 full range
    if ((vetex_x%2 == 0) && (vetex_y%2 == 0))
    {
        int j = vetex_y/2*uv_pitch+vetex_x/2;
        //u[j] = (-0.1006f*color.x - 0.3386f*color.y + 0.4392f*color.z)*256 + 128;
        //v[j] = (0.4392f*color.x - 0.3989f*color.y - 0.0403f*color.z)*256 + 128;
        //u[j] = (-173*color.x - 339*color.y + 512*color.z)*255.0f/1024.0f + 128;
        //v[j] = (512*color.x - 429*color.y - 83*color.z)*255.0f/1024.0f + 128;
        u[j] = (-38*color.x - 74*color.y + 112*color.z) + 128;
        v[j] = (112*color.x - 94*color.y - 18*color.z) + 128;
        //u[j] = (-43*color.x - 84*color.y + 127*color.z) + 128;
        //v[j] = (127*color.x - 106*color.y - 21*color.z) + 128;
    }
}

int transform_scaling_cu(const CUeglFrame& in_eglframe, const std::vector<CUeglFrame>& v_out_eglframe, std::vector<bool>& v_half)
{
     hipError_t ret;

     auto in_array = (hipArray_t)in_eglframe.frame.pArray[0];

    //get array info
    struct hipChannelFormatDesc desc;
    struct hipExtent extent;
    unsigned int flags;
    ret = hipArrayGetInfo(&desc, &extent, &flags, in_array);
    if (ret != hipSuccess) 
    {
        printf("hipArrayGetInfo fail:%d\n", ret);
        return -1;
    }

    //int w = extent.width;
    //int h = extent.height;
    //int num_channel = (desc.x + desc.y + desc.z + desc.w) / 8;
    //printf("array w:%d h:%d pitch:%d\n", w, h, y_pitch);

    //create texture obj
    struct hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = in_array;

    struct hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.filterMode = hipFilterModeLinear;
    tex_desc.readMode = hipReadModeNormalizedFloat;
    tex_desc.normalizedCoords = 1;

    hipTextureObject_t tex_obj;
    ret = hipCreateTextureObject(&tex_obj, &res_desc, &tex_desc, NULL);
    if (ret != hipSuccess) 
    {
        printf("hipCreateTextureObject fail:%d\n", ret);
        return -1;
    }
    
    for (uint32_t i = 0; i < v_out_eglframe.size(); i++)
    {
        int uv_pitch = v_out_eglframe[i].pitch/2;
        if (uv_pitch%256)
        {
            uv_pitch = (uv_pitch/256 + 1)*256;
        }

        int blocks_w = ceil(v_out_eglframe[i].width/32.0);
        int blocks_h = ceil(v_out_eglframe[i].height/16.0);
        
        //每个block的thread数目不要超过1024个
        dim3 thread_per_block(32, 16);
        dim3 blocks(blocks_w,blocks_h);
        rgba_to_yuv420_kernel<<<blocks,thread_per_block>>>(
                        tex_obj, 
                        (char*)v_out_eglframe[i].frame.pPitch[0],
                        (char*)v_out_eglframe[i].frame.pPitch[1],
                        (char*)v_out_eglframe[i].frame.pPitch[2], 
                        v_out_eglframe[i].width, 
                        v_half[i]?v_out_eglframe[i].height*2:v_out_eglframe[i].height, 
                        v_out_eglframe[i].pitch, uv_pitch);
    }

    #if 0
    if (v_out_eglframe.size() >= 2) //默认第二路为预览流了
    {
        uv_pitch = v_out_eglframe[1].pitch/2;
        if (uv_pitch%256)
        {
            uv_pitch = (uv_pitch/256 + 1)*256;
        }
        
        dim3 thread_per_block(32, 16);
        dim3 blocks(v_out_eglframe[1].width/32,v_out_eglframe[1].height/16);
        rgba_to_yuv420_kernel<<<blocks,thread_per_block>>>(
                        tex_obj, 
                        (char*)v_out_eglframe[1].frame.pPitch[0],
                        (char*)v_out_eglframe[1].frame.pPitch[1],
                        (char*)v_out_eglframe[1].frame.pPitch[2], 
                        v_out_eglframe[1].width, 
                        b_3d?v_out_eglframe[1].height*2:v_out_eglframe[1].height, 
                        v_out_eglframe[1].pitch, uv_pitch);
    }
    #endif

    hipDestroyTextureObject(tex_obj);

    return 0;
}

#if 0
int rgba_to_yuv420_cu(hipArray_t in, void** out, int y_pitch)
{
    hipError_t ret;

    //get array info
    struct hipChannelFormatDesc desc;
    struct hipExtent extent;
    unsigned int flags;
    ret = hipArrayGetInfo(&desc, &extent, &flags, in);
    if (ret != hipSuccess) 
    {
        printf("hipArrayGetInfo fail:%d\n", ret);
        return -1;
    }

    int w = extent.width;
    int h = extent.height;
    //int num_channel = (desc.x + desc.y + desc.z + desc.w) / 8;
    //printf("array w:%d h:%d pitch:%d\n", w, h, y_pitch);

    //create texture obj
    struct hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = in;

    struct hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.filterMode = hipFilterModeLinear;
    tex_desc.readMode = hipReadModeNormalizedFloat;
    tex_desc.normalizedCoords = 1;

    hipTextureObject_t tex_obj;
    ret = hipCreateTextureObject(&tex_obj, &res_desc, &tex_desc, NULL);
    if (ret != hipSuccess) 
    {
        printf("hipCreateTextureObject fail:%d\n", ret);
        return -1;
    }
    
    int uv_pitch = y_pitch/2;
    if (uv_pitch%256)
    {
        uv_pitch = (uv_pitch/256 + 1)*256;
    }

    dim3 threadsPerBlock(16, 16); //每个block的thread数目不要超过1024个
    dim3 blocks(w/16,h/16);
    rgba_to_yuv420_kernel<<<blocks,threadsPerBlock>>>(tex_obj, (char*)out[0], (char*)out[1], (char*)out[2], w, h, y_pitch, uv_pitch);

    hipDestroyTextureObject(tex_obj);

    return 0;
}
#endif


